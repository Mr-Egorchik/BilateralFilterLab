﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include "EasyBMP.h"

void writefile(float* image, int height, int width, bool gpu = false) {
    BMP output;
    output.SetSize(width, height);

    for (int i = 0; i < height; i++) {
        for (int j = 0; j < width; j++) {
            RGBApixel pixel;
            pixel.Red = image[i * width + j];
            pixel.Green = image[i * width + j];
            pixel.Blue = image[i * width + j];
            output.SetPixel(j, i, pixel);
        }
    }
    output.WriteToFile(gpu ? "output_gpu.bmp" : "output_cpu.bmp");
}

int main()
{
    BMP Image;
    Image.ReadFromFile("input.bmp");
    int height = Image.TellHeight();
    int width = Image.TellWidth();

    float* imageArray = new float[height * width];
    float* outputCPU = new float[height * width];
    float* outputGPU = new float[height * width];

    for (int j = 0; j < Image.TellHeight(); j++) {
        for (int i = 0; i < Image.TellWidth(); i++) {
            imageArray[j * width + i] = Image(i, j)->Red;
        }
    }

    writefile(imageArray, height, width);

    delete[] imageArray;
    delete[] outputCPU;
    delete[] outputGPU;

    return 0;
}